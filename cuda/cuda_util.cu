#include "cuda_util.cuh"
#include <hip/hip_runtime_api.h>

void lis::cuda::sync()
{
	checkCudaErrors(hipDeviceSynchronize());
}


void lis::cuda::peek()
{
	checkCudaErrors(hipPeekAtLastError());
}


//hipError_t lis::cuda::sync()  // for debug
//{
//	return hipDeviceSynchronize();
//}
//
//hipError_t lis::cuda::peek()
//{
//	return hipPeekAtLastError();
//}


void lis::cuda::copy
(
	void* dst,
	void* src,
	size_t count
)
{
	checkCudaErrors(hipMemcpy(dst, src, count, hipMemcpyDefault));
}

void* lis::cuda::malloc_unified
(
	size_t size
)
{
	void* ptr;
	checkCudaErrors(hipMallocManaged(&ptr, size));
	return ptr;
}

void* lis::cuda::malloc_pinned
(
	size_t size
)
{
	void* ptr;
	checkCudaErrors(hipHostMalloc(&ptr, size));
	return ptr;
}

void* lis::cuda::malloc_device
(
	size_t size
)
{
	void* ptr;
	checkCudaErrors(hipMalloc(&ptr, size));
	return ptr;
}

void lis::cuda::free_unified
(
	void* ptr
)
{
	checkCudaErrors(hipFree(ptr));
}

void lis::cuda::free_pinned
(
	void* ptr
)
{
	checkCudaErrors(hipHostFree(ptr));
}

void lis::cuda::free_device
(
	void* ptr
)
{
	checkCudaErrors(hipFree(ptr));
}

int lis::cuda::get_device()
{
	int device;
	checkCudaErrors(hipGetDevice(&device));
	return device;
}

void lis::cuda::get_device_properties
(
	hipDeviceProp_t& properties,
	int device
)
{
	checkCudaErrors(hipGetDeviceProperties(&properties, device));
}

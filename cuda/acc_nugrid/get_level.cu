#include "hip/hip_runtime.h"
#include "get_level.cuh"

__device__ int get_level(index_1D idx)
{
	return log( C(3.0) * idx + 1) / log( C(4.0) );
}
#include <hip/hip_runtime_api.h>

template<typename T>
void lis::cuda::copy_to_symbol
(
	const T& symbol,
	const void* src,
	size_t count
)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(symbol), src, count));
}

